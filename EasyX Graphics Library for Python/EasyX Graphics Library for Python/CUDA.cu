#include "hip/hip_runtime.h"
#include "CUDA.cuh"

void GPU_Device_Initialize() {
	int gpuCount = -1;
	hipGetDeviceCount(&gpuCount);
	//printf("gpuCount: %d\n", gpuCount);
	if (gpuCount <= 0)printf("ERROR: NO CUDA GPU\n");

	hipSetDevice(gpuCount - 1);
}

//__global__ void rotate_X(float *PointStack_X_GPU, float angle, POINT Base) {
//	int i = threadIdx.x;
//	PointStack_X_GPU[i] = PointStack_X_GPU[i] * cos(angle) - PointStack_X_GPU[i] * sin(angle) + Base.x * (1 - cos(angle)) + Base.y * sin(angle);
//}
//__global__ void rotate_Y(float *PointStack_Y_GPU, float angle, POINT Base) {
//	int i = threadIdx.x;
//	PointStack_Y_GPU[i] = PointStack_Y_GPU[i] * sin(angle) + PointStack_Y_GPU[i] * cos(angle) + Base.y * (1 - cos(angle)) - Base.x * sin(angle);
//}
//
//float **GPU_rotate(vector<float> PointStack_X, vector<float> PointStack_Y, float angle, POINT Base) {
//	GPU_Device_Initialize();
//
//	//init data;
//	float *PointStack_X_CPU = new float[PointStack_X.size()];
//	float *PointStack_Y_CPU = new float[PointStack_Y.size()];
//
//	float *PointStack_X_GPU = nullptr;
//	float *PointStack_Y_GPU = nullptr;
//
//	for (int i = 0; i < PointStack_X.size(); ++i) PointStack_X_CPU[i] = PointStack_X[i];
//	for (int i = 0; i < PointStack_Y.size(); ++i) PointStack_Y_CPU[i] = PointStack_Y[i];
//
//	//new space;
//	hipMalloc((void **)&PointStack_X_GPU, PointStack_X.size() * sizeof(float));
//	hipMalloc((void **)&PointStack_Y_GPU, PointStack_Y.size() * sizeof(float));
//
//	//copy data CPU -> GPU;
//	hipMemcpy(PointStack_X_GPU, PointStack_X_CPU, PointStack_X.size() * sizeof(float), hipMemcpyHostToDevice);
//	hipMemcpy(PointStack_Y_GPU, PointStack_Y_CPU, PointStack_Y.size() * sizeof(float), hipMemcpyHostToDevice);
//
//	//do;
//	int threadnum = 1024;
//	int blocknum = 1024;
//
//	rotate_X << <blocknum, threadnum >> > (PointStack_X_GPU, angle, Base);
//	rotate_Y << <blocknum, threadnum >> > (PointStack_Y_GPU, angle, Base);
//
//	//copy data GPU -> CPU;
//	hipMemcpy(PointStack_X_CPU, PointStack_X_GPU, PointStack_X.size() * sizeof(float), hipMemcpyDeviceToHost);
//	hipMemcpy(PointStack_Y_CPU, PointStack_Y_GPU, PointStack_Y.size() * sizeof(float), hipMemcpyDeviceToHost);
//
//	float **PointStack_GPU = new float *[PointStack_X.size()] { NULL };
//	for (int i = 0; i < PointStack_X.size(); ++i) {
//		PointStack_GPU[i] = new float[2] {NULL};
//
//		PointStack_GPU[i][0] = PointStack_X_CPU[i];
//		PointStack_GPU[i][1] = PointStack_Y_CPU[i];
//	}
//
//	//free;
//	hipFree(PointStack_X_GPU); delete[]PointStack_X_CPU;
//	hipFree(PointStack_Y_GPU); delete[]PointStack_Y_CPU;
//
//	/*for (int i = 0; i < PointStack_X.size(); ++i)cout << PointStack_X_CPU[i] << endl;
//	cout << endl;
//	for (int i = 0; i < PointStack_Y.size(); ++i)cout << PointStack_Y_CPU[i] << endl;*/
//
//	//hipDeviceReset();
//
//	return PointStack_GPU;
//}

__global__ void rotate(float **PointStack_GPU, float angle, POINT Base) {
	int i = threadIdx.x;
}

float **GPU_rotate(float **PointStack, int num, float angle, POINT Base) {
	GPU_Device_Initialize();

	//init data;
	float **PointStack_GPU = nullptr;
	float **PointStack_CPU = nullptr;

	//new space;
	size_t pitch;
	hipMallocPitch((void **)&PointStack_GPU, &pitch, width * sizeof(float), height);
	//hipMallocPitch((void **)&PointStack_GPU, num * sizeof(float));

	//copy data CPU -> GPU;
	hipMemcpy(PointStack_GPU, PointStack, num * sizeof(float), hipMemcpyHostToDevice);

	//do;
	rotate << <1, 1024 >> > (PointStack_GPU, angle, Base);

	//copy data GPU -> CPU;
	hipMemcpy(PointStack_CPU, PointStack_GPU, num * sizeof(float), hipMemcpyDeviceToHost);

	//free;
	hipFree(PointStack_GPU);

	return nullptr;
}