#include "hip/hip_runtime.h"
#include "CUDA.cuh"

__global__ void helloGPU() {
	printf("Hello from the GPU.\n");
}

void TEST() {
	helloGPU << <100, 100 >> > ();
	hipDeviceSynchronize();
}