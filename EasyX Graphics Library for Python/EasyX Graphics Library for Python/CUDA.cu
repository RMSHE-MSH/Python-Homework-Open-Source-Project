#include "hip/hip_runtime.h"
#include "CUDA.cuh"

void GPU_Device_Initialize() {
	int gpuCount = -1;
	hipGetDeviceCount(&gpuCount);
	//printf("gpuCount: %d\n", gpuCount);
	if (gpuCount <= 0)printf("ERROR: NO CUDA GPU\n");

	hipSetDevice(gpuCount - 1);
}

//__global__ void rotate_X(float *PointStack_X_GPU, float angle, POINT Base) {
//	int i = threadIdx.x;
//	PointStack_X_GPU[i] = PointStack_X_GPU[i] * cos(angle) - PointStack_X_GPU[i] * sin(angle) + Base.x * (1 - cos(angle)) + Base.y * sin(angle);
//}
//__global__ void rotate_Y(float *PointStack_Y_GPU, float angle, POINT Base) {
//	int i = threadIdx.x;
//	PointStack_Y_GPU[i] = PointStack_Y_GPU[i] * sin(angle) + PointStack_Y_GPU[i] * cos(angle) + Base.y * (1 - cos(angle)) - Base.x * sin(angle);
//}
//
//float **GPU_rotate(vector<float> PointStack_X, vector<float> PointStack_Y, float angle, POINT Base) {
//	GPU_Device_Initialize();
//
//	//init data;
//	float *PointStack_X_CPU = new float[PointStack_X.size()];
//	float *PointStack_Y_CPU = new float[PointStack_Y.size()];
//
//	float *PointStack_X_GPU = nullptr;
//	float *PointStack_Y_GPU = nullptr;
//
//	for (int i = 0; i < PointStack_X.size(); ++i) PointStack_X_CPU[i] = PointStack_X[i];
//	for (int i = 0; i < PointStack_Y.size(); ++i) PointStack_Y_CPU[i] = PointStack_Y[i];
//
//	//new space;
//	hipMalloc((void **)&PointStack_X_GPU, PointStack_X.size() * sizeof(float));
//	hipMalloc((void **)&PointStack_Y_GPU, PointStack_Y.size() * sizeof(float));
//
//	//copy data CPU -> GPU;
//	hipMemcpy(PointStack_X_GPU, PointStack_X_CPU, PointStack_X.size() * sizeof(float), hipMemcpyHostToDevice);
//	hipMemcpy(PointStack_Y_GPU, PointStack_Y_CPU, PointStack_Y.size() * sizeof(float), hipMemcpyHostToDevice);
//
//	//do;
//	int threadnum = 1024;
//	int blocknum = 1024;
//
//	rotate_X << <blocknum, threadnum >> > (PointStack_X_GPU, angle, Base);
//	rotate_Y << <blocknum, threadnum >> > (PointStack_Y_GPU, angle, Base);
//
//	//copy data GPU -> CPU;
//	hipMemcpy(PointStack_X_CPU, PointStack_X_GPU, PointStack_X.size() * sizeof(float), hipMemcpyDeviceToHost);
//	hipMemcpy(PointStack_Y_CPU, PointStack_Y_GPU, PointStack_Y.size() * sizeof(float), hipMemcpyDeviceToHost);
//
//	float **PointStack_GPU = new float *[PointStack_X.size()] { NULL };
//	for (int i = 0; i < PointStack_X.size(); ++i) {
//		PointStack_GPU[i] = new float[2] {NULL};
//
//		PointStack_GPU[i][0] = PointStack_X_CPU[i];
//		PointStack_GPU[i][1] = PointStack_Y_CPU[i];
//	}
//
//	//free;
//	hipFree(PointStack_X_GPU); delete[]PointStack_X_CPU;
//	hipFree(PointStack_Y_GPU); delete[]PointStack_Y_CPU;
//
//	/*for (int i = 0; i < PointStack_X.size(); ++i)cout << PointStack_X_CPU[i] << endl;
//	cout << endl;
//	for (int i = 0; i < PointStack_Y.size(); ++i)cout << PointStack_Y_CPU[i] << endl;*/
//
//	//hipDeviceReset();
//
//	return PointStack_GPU;
//}

__global__ void rotate_X(float *PointStack_X_GPU, float *PointStack_Y_GPU, float angle, POINT Base) {
	int i = threadIdx.x;

	//X;
	PointStack_X_GPU[i] = PointStack_X_GPU[i] * cos(angle) - PointStack_Y_GPU[i] * sin(angle) + Base.x * (1 - cos(angle)) + Base.y * sin(angle);
}

__global__ void rotate_Y(float *PointStack_X_GPU, float *PointStack_Y_GPU, float angle, POINT Base) {
	int i = threadIdx.x;

	//Y;
	PointStack_Y_GPU[i] = PointStack_X_GPU[i] * sin(angle) + PointStack_Y_GPU[i] * cos(angle) + Base.y * (1 - cos(angle)) - Base.x * sin(angle);
}

float *GPU_rotate(float *PointStack_CPU, int PointStackSize, float angle, POINT Base) {
	GPU_Device_Initialize();

	//init data;
	float *PointStack_X_GPU = nullptr;
	float *PointStack_Y_GPU = nullptr;

	float *GPU_X_Result = new float[PointStackSize] {};
	float *GPU_Y_Result = new float[PointStackSize] {};

	float *PointStack_X_CPU = new float[PointStackSize] {};
	float *PointStack_Y_CPU = new float[PointStackSize] {};

	int X = 0; for (int i = 0; i < 2 * PointStackSize; i += 2, ++X) PointStack_X_CPU[X] = PointStack_CPU[i];
	int Y = 0; for (int i = 1; i < 2 * PointStackSize; i += 2, ++Y) PointStack_Y_CPU[Y] = PointStack_CPU[i];

	//�����Դ�ռ�;
	hipMalloc((void **)&PointStack_X_GPU, PointStackSize * sizeof(float));
	hipMalloc((void **)&PointStack_Y_GPU, PointStackSize * sizeof(float));

	//copy data CPU -> GPU;
	hipMemcpy(PointStack_X_GPU, PointStack_X_CPU, PointStackSize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(PointStack_Y_GPU, PointStack_Y_CPU, PointStackSize * sizeof(float), hipMemcpyHostToDevice);

	//��������;
	int BlockNum = 1;
	int ThreadNum = PointStackSize;

	//��ʼ���м���;
	rotate_X << <BlockNum, ThreadNum >> > (PointStack_X_GPU, PointStack_Y_GPU, angle, Base);
	rotate_Y << <BlockNum, ThreadNum >> > (PointStack_X_GPU, PointStack_Y_GPU, angle, Base);

	//copy data GPU -> CPU;
	hipMemcpy(GPU_X_Result, PointStack_X_GPU, PointStackSize * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(GPU_Y_Result, PointStack_Y_GPU, PointStackSize * sizeof(float), hipMemcpyDeviceToHost);

	float *GPU_Result = new float[2 * PointStackSize] {};

	int _X = 0; for (int i = 0; i < PointStackSize; ++i, _X += 2) GPU_Result[_X] = GPU_X_Result[i];
	int _Y = 1; for (int i = 0; i < PointStackSize; ++i, _Y += 2) GPU_Result[_Y] = GPU_Y_Result[i];

	//�ͷ��Դ�;
	hipFree(PointStack_X_GPU);
	hipFree(PointStack_Y_GPU);

	return GPU_Result;
}